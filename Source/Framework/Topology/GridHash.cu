#include "hip/hip_runtime.h"
#pragma once
#include "GridHash.h"
#include "Core/Utility.h"
#include "Primitive3D.h"

namespace PhysIKA {

	__constant__ int offset[27][3] = { 0, 0, 0,
		0, 0, 1,
		0, 1, 0,
		1, 0, 0,
		0, 0, -1,
		0, -1, 0,
		-1, 0, 0,
		0, 1, 1,
		0, 1, -1,
		0, -1, 1,
		0, -1, -1,
		1, 0, 1,
		1, 0, -1,
		-1, 0, 1,
		-1, 0, -1,
		1, 1, 0,
		1, -1, 0,
		-1, 1, 0,
		-1, -1, 0,
		1, 1, 1,
		1, 1, -1,
		1, -1, 1,
		-1, 1, 1,
		1, -1, -1,
		-1, 1, -1,
		-1, -1, 1,
		-1, -1, -1
	};

	template<typename TDataType>
	GridHash<TDataType>::GridHash()
	{
	}

	template<typename TDataType>
	GridHash<TDataType>::~GridHash()
	{
	}

	template<typename TDataType>
	void GridHash<TDataType>::setSpace(Real _h, Coord _lo, Coord _hi)
	{
		release();

		int padding = 2;
		ds = _h;
		lo = _lo - padding * ds;

		Coord nSeg = (_hi - _lo) / ds;

		nx = ceil(nSeg[0]) + 1 + 2 * padding;
		ny = ceil(nSeg[1]) + 1 + 2 * padding;
		nz = ceil(nSeg[2]) + 1 + 2 * padding;
		hi = lo + Coord(nx, ny, nz) * ds;

		num = nx * ny * nz;

		//		npMax = 128;

		cuSafeCall(hipMalloc((void**)&counter, num * sizeof(int)));
		cuSafeCall(hipMalloc((void**)&index, num * sizeof(int)));

		if (m_reduce != nullptr)
		{
			delete m_reduce;
		}

		m_reduce = Reduction<int>::Create(num);
		if (multi_grid)
			initializeMultiLevel();
	}
	template<typename TDataType>
	void GridHash<TDataType>::initializeMultiLevel()
	{
		release();
		int level = 0;
		Coord tmp = hi - lo;
		Real maxx = max(tmp[0], tmp[1]);
		maxx = max(maxx, tmp[2]);
		int padding = 2;

		while ((1 << level) * ds < maxx && level < 10)
		{
			Real ds_i = (1 << level) * ds;
			Coord nSeg = (hi - lo) / ds_i;

			nx = ceil(nSeg[0]) + 1 + 2 * padding;
			ny = ceil(nSeg[1]) + 1 + 2 * padding;
			nz = ceil(nSeg[2]) + 1 + 2 * padding;

			int num_i = nx * ny * nz;
			nx_list[level] = nx;
			ny_list[level] = ny;
			nz_list[level] = nz;

			prefix[level] = num_i;
			if (level >= 1)
				prefix[level] += prefix[level - 1];
			level++;
		};
		level--;
		num = prefix[level];
		maxlevel = level;

		cuSafeCall(hipMalloc((void**)&counter, num * sizeof(int)));
		cuSafeCall(hipMalloc((void**)&index, num * sizeof(int)));
		if (m_reduce != nullptr)
		{
			delete m_reduce;
		}

		m_reduce = Reduction<int>::Create(num);

	}

	template<typename TDataType>
	__global__ void K_CalculateParticleNumber(GridHash<TDataType> hash, Array<typename TDataType::Coord> pos)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= pos.size()) return;

		int gId = hash.getIndex(pos[pId]);

		if (gId != INVALID)
			atomicAdd(&(hash.index[gId]), 1);
	}


	template<typename TDataType>
	__global__ void K_AddTriNumber(GridHash<TDataType> hash, Array<typename TopologyModule::Triangle> tri, Array<typename TDataType::Coord> pos)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= tri.size()) return;
		//printf("TRISIZE:%.3lf %.3lf %.3lf\n",hash.lo[0], hash.lo[1], hash.lo[2]);
		/*
		int gId1 = hash.getIndex(pos[tri[pId][0]]);
		int gId2 = hash.getIndex(pos[tri[pId][1]]);
		int gId3 = hash.getIndex(pos[tri[pId][2]]);
		*/
		Real ds = hash.ds;
		//Coord3D lo = hash.lo;
		//Coord3D hi = hash.hi;

		int i0 = floor((pos[tri[pId][0]][0] - hash.lo[0]) / hash.ds);
		int j0 = floor((pos[tri[pId][0]][1] - hash.lo[1]) / hash.ds);
		int k0 = floor((pos[tri[pId][0]][2] - hash.lo[2]) / hash.ds);

		int i1 = floor((pos[tri[pId][1]][0] - hash.lo[0]) / hash.ds);
		int j1 = floor((pos[tri[pId][1]][1] - hash.lo[1]) / hash.ds);
		int k1 = floor((pos[tri[pId][1]][2] - hash.lo[2]) / hash.ds);

		int i2 = floor((pos[tri[pId][2]][0] - hash.lo[0]) / hash.ds);
		int j2 = floor((pos[tri[pId][2]][1] - hash.lo[1]) / hash.ds);
		int k2 = floor((pos[tri[pId][2]][2] - hash.lo[2]) / hash.ds);

		int imin = i0 < i1 ? i0 : i1;
		imin = i2 < imin ? i2 : imin;
		int imax = i0 > i1 ? i0 : i1;
		imax = i2 > imax ? i2 : imax;

		int jmin = j0 < j1 ? j0 : j1;
		jmin = j2 < jmin ? j2 : jmin;
		int jmax = j0 > j1 ? j0 : j1;
		jmax = j2 > jmax ? j2 : jmax;

		int kmin = k0 < k1 ? k0 : k1;
		kmin = k2 < kmin ? k2 : kmin;
		int kmax = k0 > k1 ? k0 : k1;
		kmax = k2 > kmax ? k2 : kmax;

		imin--; jmin--; kmin--;
		imax++; jmax++; kmax++;

		int addi, addj, addk;
		addi = int(sqrt((Real)imax - (Real)imin + 1));
		addj = int(sqrt((Real)jmax - (Real)jmin + 1));
		addk = int(sqrt((Real)kmax - (Real)kmin + 1));

		Triangle3D t3d = Triangle3D(pos[tri[pId][0]], pos[tri[pId][1]], pos[tri[pId][2]]);
		//printf("%d %d %d\n",addi,addj,addk);
		for (int li = imin; li <= imax; li += 1)
			for (int lj = jmin; lj <= jmax; lj += 1)
				for (int lk = kmin; lk <= kmax; lk += 1)
				{

					int i = li, j = lj, k = lk;
					Coord3D ABP11 = Coord3D(i * ds + hash.lo[0] - 0.1 * ds * 10.0,
						j * ds + hash.lo[1] - 0.1 * ds * 10.0,
						k * ds + hash.lo[2] - 0.1 * ds * 10.0);
					Coord3D ABP22 = Coord3D(i * ds + ds + hash.lo[0] + 0.1 * ds * 10.0,
						j * ds + ds + hash.lo[1] + 0.1 * ds * 10.0,
						k * ds + ds + hash.lo[2] + 0.1 * ds * 10.0);
					AlignedBox3D AABB2 = AlignedBox3D(ABP11, ABP22);

					if (AABB2.meshInsert(t3d))
					{
						int gId = hash.getIndex(i, j, k);
						if (gId != INVALID)
							atomicAdd(&(hash.index[gId]), 1);

					}

				}

	}

	template<typename TDataType>
	__global__ void K_Multi_AddTriNumber(GridHash<TDataType> hash, Array<typename TopologyModule::Triangle> tri, Array<typename TDataType::Coord> pos)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= tri.size()) return;

		Real ds = hash.ds;
		int level = -1;
		int i0, j0, k0;
		int i1, j1, k1;
		int i2, j2, k2;
		int imin, imax, jmin, jmax, kmin, kmax;
		do {

			level++;
			ds = hash.ds * (Real)(1 << level);

			i0 = floor((pos[tri[pId][0]][0] - hash.lo[0]) / ds);
			j0 = floor((pos[tri[pId][0]][1] - hash.lo[1]) / ds);
			k0 = floor((pos[tri[pId][0]][2] - hash.lo[2]) / ds);

			i1 = floor((pos[tri[pId][1]][0] - hash.lo[0]) / ds);
			j1 = floor((pos[tri[pId][1]][1] - hash.lo[1]) / ds);
			k1 = floor((pos[tri[pId][1]][2] - hash.lo[2]) / ds);

			i2 = floor((pos[tri[pId][2]][0] - hash.lo[0]) / ds);
			j2 = floor((pos[tri[pId][2]][1] - hash.lo[1]) / ds);
			k2 = floor((pos[tri[pId][2]][2] - hash.lo[2]) / ds);

			imin = i0 < i1 ? i0 : i1;
			imin = i2 < imin ? i2 : imin;
			imax = i0 > i1 ? i0 : i1;
			imax = i2 > imax ? i2 : imax;

			jmin = j0 < j1 ? j0 : j1;
			jmin = j2 < jmin ? j2 : jmin;
			jmax = j0 > j1 ? j0 : j1;
			jmax = j2 > jmax ? j2 : jmax;

			kmin = k0 < k1 ? k0 : k1;
			kmin = k2 < kmin ? k2 : kmin;
			kmax = k0 > k1 ? k0 : k1;
			kmax = k2 > kmax ? k2 : kmax;

			imin--; jmin--; kmin--;
			imax++; jmax++; kmax++;

		} while ((imax - imin + 1) * (jmax - jmin + 1) * (kmax - kmin + 1) > 150 && level < hash.maxlevel);


		//ds = hash.ds;
		Triangle3D t3d = Triangle3D(pos[tri[pId][0]], pos[tri[pId][1]], pos[tri[pId][2]]);

		for (int li = imin; li <= imax; li += 1)
			for (int lj = jmin; lj <= jmax; lj += 1)
				for (int lk = kmin; lk <= kmax; lk += 1)
				{
					int i = li, j = lj, k = lk;
					Coord3D ABP11 = Coord3D(i * ds + hash.lo[0] - 0.1 * hash.ds * 10.0,
						j * ds + hash.lo[1] - 0.1 * hash.ds * 10.0,
						k * ds + hash.lo[2] - 0.1 * hash.ds * 10.0);
					Coord3D ABP22 = Coord3D(i * ds + ds + hash.lo[0] + 0.1 * hash.ds * 10.0,
						j * ds + ds + hash.lo[1] + 0.1 * hash.ds * 10.0,
						k * ds + ds + hash.lo[2] + 0.1 * hash.ds * 10.0);
					AlignedBox3D AABB2 = AlignedBox3D(ABP11, ABP22);

					if (AABB2.meshInsert(t3d))
					{
						int gId = hash.getIndex(i, j, k, level);
						//printf("^^^^^^^^^^^^^^^^^^^^^^^^^^^ %d %d %d\n", gId, level, INVALID);
						if (gId != INVALID)
							atomicAdd(&(hash.index[gId]), 1);

					}


				}

	}

	template<typename TDataType>
	__global__ void K_ConstructHashTable(GridHash<TDataType> hash, Array<typename TDataType::Coord> pos)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= pos.size()) return;

		int gId = hash.getIndex(pos[pId]);

		if (gId < 0) return;

		int index = atomicAdd(&(hash.counter[gId]), 1);
		// 		index = index < hash.npMax - 1 ? index : hash.npMax - 1;
		// 		hash.ids[gId * hash.npMax + index] = pId;
		hash.ids[hash.index[gId] + index] = pId;
	}


	template<typename TDataType>
	__global__ void K_AddTriElement(GridHash<TDataType> hash, Array<typename TopologyModule::Triangle> tri, Array<typename TDataType::Coord> pos)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= tri.size()) return;
		Real ds = hash.ds;
		//Coord3D lo = hash.lo;
		//Coord3D hi = hash.hi;
		int i0 = floor((pos[tri[pId][0]][0] - hash.lo[0]) / hash.ds);
		int j0 = floor((pos[tri[pId][0]][1] - hash.lo[1]) / hash.ds);
		int k0 = floor((pos[tri[pId][0]][2] - hash.lo[2]) / hash.ds);

		int i1 = floor((pos[tri[pId][1]][0] - hash.lo[0]) / hash.ds);
		int j1 = floor((pos[tri[pId][1]][1] - hash.lo[1]) / hash.ds);
		int k1 = floor((pos[tri[pId][1]][2] - hash.lo[2]) / hash.ds);

		int i2 = floor((pos[tri[pId][2]][0] - hash.lo[0]) / hash.ds);
		int j2 = floor((pos[tri[pId][2]][1] - hash.lo[1]) / hash.ds);
		int k2 = floor((pos[tri[pId][2]][2] - hash.lo[2]) / hash.ds);

		int imin = i0 < i1 ? i0 : i1;
		imin = i2 < imin ? i2 : imin;
		int imax = i0 > i1 ? i0 : i1;
		imax = i2 > imax ? i2 : imax;

		int jmin = j0 < j1 ? j0 : j1;
		jmin = j2 < jmin ? j2 : jmin;
		int jmax = j0 > j1 ? j0 : j1;
		jmax = j2 > jmax ? j2 : jmax;

		int kmin = k0 < k1 ? k0 : k1;
		kmin = k2 < kmin ? k2 : kmin;
		int kmax = k0 > k1 ? k0 : k1;
		kmax = k2 > kmax ? k2 : kmax;
		imin--; jmin--; kmin--;
		imax++; jmax++; kmax++;
		int addi, addj, addk;
		addi = int(sqrt((Real)imax - (Real)imin + 1));
		addj = int(sqrt((Real)jmax - (Real)jmin + 1));
		addk = int(sqrt((Real)kmax - (Real)kmin + 1));

		Triangle3D t3d = Triangle3D(pos[tri[pId][0]], pos[tri[pId][1]], pos[tri[pId][2]]);


		for (int li = imin; li <= imax; li += 1)
			for (int lj = jmin; lj <= jmax; lj += 1)
				for (int lk = kmin; lk <= kmax; lk += 1)
				{

					int i = li, j = lj, k = lk;
					Coord3D ABP11 = Coord3D(i * ds + hash.lo[0] - 0.1 * ds * 10.0,
						j * ds + hash.lo[1] - 0.1 * ds * 10.0,
						k * ds + hash.lo[2] - 0.1 * ds * 10.0);
					Coord3D ABP22 = Coord3D(i * ds + ds + hash.lo[0] + 0.1 * ds * 10.0,
						j * ds + ds + hash.lo[1] + 0.1 * ds * 10.0,
						k * ds + ds + hash.lo[2] + 0.1 * ds * 10.0);
					AlignedBox3D AABB2 = AlignedBox3D(ABP11, ABP22);

					if (AABB2.meshInsert(t3d))
					{
						int gId = hash.getIndex(i, j, k);

						if (gId != INVALID)
						{
							int index = atomicAdd(&(hash.counter[gId]), 1);
							hash.ids[hash.index[gId] + index] = -pId - 1;
						}

					}
				}


	}

	template<typename TDataType>
	__global__ void K_Multi_AddTriElement(GridHash<TDataType> hash, Array<typename TopologyModule::Triangle> tri, Array<typename TDataType::Coord> pos)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= tri.size()) return;

		Real ds = hash.ds;
		int level = -1;
		int i0, j0, k0;
		int i1, j1, k1;
		int i2, j2, k2;
		int imin, imax, jmin, jmax, kmin, kmax;
		do {

			level++;
			ds = hash.ds * (Real)(1 << level);

			i0 = floor((pos[tri[pId][0]][0] - hash.lo[0]) / ds);
			j0 = floor((pos[tri[pId][0]][1] - hash.lo[1]) / ds);
			k0 = floor((pos[tri[pId][0]][2] - hash.lo[2]) / ds);

			i1 = floor((pos[tri[pId][1]][0] - hash.lo[0]) / ds);
			j1 = floor((pos[tri[pId][1]][1] - hash.lo[1]) / ds);
			k1 = floor((pos[tri[pId][1]][2] - hash.lo[2]) / ds);

			i2 = floor((pos[tri[pId][2]][0] - hash.lo[0]) / ds);
			j2 = floor((pos[tri[pId][2]][1] - hash.lo[1]) / ds);
			k2 = floor((pos[tri[pId][2]][2] - hash.lo[2]) / ds);

			imin = i0 < i1 ? i0 : i1;
			imin = i2 < imin ? i2 : imin;
			imax = i0 > i1 ? i0 : i1;
			imax = i2 > imax ? i2 : imax;

			jmin = j0 < j1 ? j0 : j1;
			jmin = j2 < jmin ? j2 : jmin;
			jmax = j0 > j1 ? j0 : j1;
			jmax = j2 > jmax ? j2 : jmax;

			kmin = k0 < k1 ? k0 : k1;
			kmin = k2 < kmin ? k2 : kmin;
			kmax = k0 > k1 ? k0 : k1;
			kmax = k2 > kmax ? k2 : kmax;

			imin--; jmin--; kmin--;
			imax++; jmax++; kmax++;

		} while ((imax - imin + 1) * (jmax - jmin + 1) * (kmax - kmin + 1) > 150 && level < hash.maxlevel);

		//	ds = hash.ds;

		Triangle3D t3d = Triangle3D(pos[tri[pId][0]], pos[tri[pId][1]], pos[tri[pId][2]]);
		for (int li = imin; li <= imax; li += 1)
			for (int lj = jmin; lj <= jmax; lj += 1)
				for (int lk = kmin; lk <= kmax; lk += 1)
				{
					int i = li, j = lj, k = lk;
					Coord3D ABP11 = Coord3D(i * ds + hash.lo[0] - 0.1 * hash.ds * 10.0,
						j * ds + hash.lo[1] - 0.1 * hash.ds * 10.0,
						k * ds + hash.lo[2] - 0.1 * hash.ds * 10.0);
					Coord3D ABP22 = Coord3D(i * ds + ds + hash.lo[0] + 0.1 * hash.ds * 10.0,
						j * ds + ds + hash.lo[1] + 0.1 * hash.ds * 10.0,
						k * ds + ds + hash.lo[2] + 0.1 * hash.ds * 10.0);
					AlignedBox3D AABB2 = AlignedBox3D(ABP11, ABP22);

					if (AABB2.meshInsert(t3d))
					{
						int gId = hash.getIndex(i, j, k, level);
						if (gId != INVALID)
						{
							int index = atomicAdd(&(hash.counter[gId]), 1);
							hash.ids[hash.index[gId] + index] = -pId - 1;
						}

					}


				}

	}
	template<typename TDataType>
	void GridHash<TDataType>::construct(DeviceArray<Coord>& pos)
	{
		clear();

		dim3 pDims = int(ceil(pos.size() / BLOCK_SIZE + 0.5f));

		K_CalculateParticleNumber << <pDims, BLOCK_SIZE >> > (*this, pos);
		particle_num = m_reduce->accumulate(index, num);

		if (m_scan == nullptr)
		{
			m_scan = new Scan();
		}
		m_scan->exclusive(index, num);

		if (ids != nullptr)
		{
			cuSafeCall(hipFree(ids));
		}
		cuSafeCall(hipMalloc((void**)&ids, particle_num * sizeof(int)));

		//		std::cout << "Particle number: " << particle_num << std::endl;

		K_ConstructHashTable << <pDims, BLOCK_SIZE >> > (*this, pos);
		cuSynchronize();
	}


	template<typename TDataType>
	void GridHash<TDataType>::construct(DeviceArray<Coord>& pos, DeviceArray<Triangle>& tri, DeviceArray<Coord>& Tri_pos)
	{
		clear();

		dim3 pDims = int(ceil(pos.size() / BLOCK_SIZE + 0.5f));
		dim3 pDimsTri = int(ceil(tri.size() / BLOCK_SIZE + 0.5f));

		//	K_CalculateParticleNumber << <pDims, BLOCK_SIZE >> > (*this, pos);
		//	cuSynchronize();
		if (!multi_grid)
			K_AddTriNumber << <pDimsTri, BLOCK_SIZE >> > (*this, tri, Tri_pos);
		else
			K_Multi_AddTriNumber << <pDimsTri, BLOCK_SIZE >> > (*this, tri, Tri_pos);

		cuSynchronize();

		particle_num = m_reduce->accumulate(index, num);

		if (m_scan == nullptr)
		{
			m_scan = new Scan();
		}
		m_scan->exclusive(index, num);

		if (ids != nullptr)
		{
			cuSafeCall(hipFree(ids));
		}
		cuSafeCall(hipMalloc((void**)&ids, particle_num * sizeof(int)));


		//	K_ConstructHashTable << <pDims, BLOCK_SIZE >> > (*this, pos);
		//	cuSynchronize();
		if (!multi_grid)
			K_AddTriElement << <pDimsTri, BLOCK_SIZE >> > (*this, tri, Tri_pos);
		else
			K_Multi_AddTriElement << <pDimsTri, BLOCK_SIZE >> > (*this, tri, Tri_pos);
		cuSynchronize();
	}

	template<typename TDataType>
	void GridHash<TDataType>::clear()
	{
		cuSafeCall(hipMemset(counter, 0, num * sizeof(int)));
		cuSafeCall(hipMemset(index, 0, num * sizeof(int)));
	}

	template<typename TDataType>
	void GridHash<TDataType>::release()
	{
		if (counter != nullptr)
			cuSafeCall(hipFree(counter));

		if (ids != nullptr)
			cuSafeCall(hipFree(ids));

		if (index != nullptr)
			cuSafeCall(hipFree(index));

		// 		if (m_scan != nullptr)
		// 		{
		// 			delete m_scan;
		// 		}
	}
}